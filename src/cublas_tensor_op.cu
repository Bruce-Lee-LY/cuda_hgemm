// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 20:42:28 on Sun, Feb 12, 2023
//
// Description: cublas tensor op hgemm

#include "common.h"

hipblasHandle_t getCublasTensorOpHandle() {
    hipblasHandle_t handle = nullptr;
    HGEMM_CHECK_CUBLAS_ERROR(hipblasCreate(&handle));
    HGEMM_CHECK_CUBLAS_ERROR(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    return handle;
}

void cublasTensorOp(half *A, half *B, half *C, size_t M, size_t N, size_t K) {
    static hipblasHandle_t handle = getCublasTensorOpHandle();
    static half alpha = 1.0;
    static half beta = 0.0;

    HGEMM_CHECK_CUBLAS_ERROR(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F, K, A,
                                          HIP_R_16F, K, &beta, C, HIP_R_16F, N, HIPBLAS_COMPUTE_16F,
                                          CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}
